#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256
#define MAX_PARTICLES_PER_BIN 32

extern double size;
//
//  benchmarking program
//

__device__ __forceinline__ void apply_force_gpu(particle_t &particle, const particle_t &neighbor)  // CHANGE: pass by reference
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if (r2 > cutoff * cutoff)
      return;
   
  r2 = fmax(r2, min_r * min_r);
  double r = sqrt(r2);
  //
  //  very simple short-range repulsive force
  //
  double coef = (1 - cutoff / r) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* __restrict__ particles, int n, int* __restrict__ bins, int* __restrict__ bin_counts, double bin_size, int bins_per_row)
{
  // Get thread (particle) ID
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid >= n) return;

  particle_t &p = particles[tid];
  p.ax = p.ay = 0;

  int bin_x = p.x / bin_size;
  int bin_y = p.y / bin_size;
  if (bin_x < 0 || bin_x >= bins_per_row || bin_y < 0 || bin_y >= bins_per_row) return;

    int bin_index = bin_y * bins_per_row + bin_x;

    int idx = atomicAdd(&bin_counts[bin_index], 1);
    if (idx < MAX_PARTICLES_PER_BIN)
        bins[bin_index * MAX_PARTICLES_PER_BIN + idx] = tid;



  for (int dx = -1; dx <= 1; dx++)
    {
        for (int dy = -1; dy <= 1; dy++)
        {
            int nbx = bin_x + dx;
            int nby = bin_y + dy;
            if (nbx < 0 || nbx >= bins_per_row || nby < 0 || nby >= bins_per_row)
                continue;

            int neighbor_bin = nby * bins_per_row + nbx;
            int count = bin_counts[neighbor_bin];

            for (int j = 0; j < count; j++)
            {
                int neighbor_idx = bins[neighbor_bin * MAX_PARTICLES_PER_BIN + j];
                if (neighbor_idx == tid) continue;
                apply_force_gpu(p, particles[neighbor_idx]);
            }
        }
    }
 
}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

__global__ void reset_bin_counts(int* d_bin_counts, int num_bins) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_bins)
        d_bin_counts[tid] = 0;
}

int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );
    
    double bin_size = cutoff;
    int bins_per_row = ceil(size / bin_size);
    int num_bins = bins_per_row * bins_per_row;

    int *d_bins;
    int *d_bin_counts;
    hipMalloc(&d_bins, num_bins * MAX_PARTICLES_PER_BIN * sizeof(int));
    hipMalloc(&d_bin_counts, num_bins * sizeof(int));

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int bin_blks = (num_bins + NUM_THREADS - 1) / NUM_THREADS;

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //
        reset_bin_counts<<<bin_blks, NUM_THREADS>>>(d_bin_counts, num_bins);
        compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n, d_bins, d_bin_counts, bin_size, bins_per_row);
        
        //
        //  move particles
        //
        move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    hipFree(d_bins);
    hipFree(d_bin_counts);
    if( fsave )
        fclose( fsave );
    
    return 0;
}


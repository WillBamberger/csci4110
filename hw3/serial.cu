#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "common.h"

#define MAX_PARTICLES_PER_BIN 64
extern double size;

//
//  benchmarking program
//
int main( int argc, char **argv )
{    
    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    set_size( n );
    init_particles( n, particles );
    
    double bin_size = cutoff;
    int bins_per_row = ceil(size / bin_size);
    int num_bins = bins_per_row * bins_per_row;

    particle_t **bins = (particle_t**) malloc(num_bins * sizeof(particle_t*));
    int *bin_counts = (int*) malloc(num_bins * sizeof(int));
    for (int i = 0; i < num_bins; i++) {
        bins[i] = (particle_t*) malloc(MAX_PARTICLES_PER_BIN * sizeof(particle_t));
        bin_counts[i] = 0;
    }

    //
    //  simulate a number of time steps
    //
    double simulation_time = read_timer( );
    for( int step = 0; step < NSTEPS; step++ )
    {
        for (int i = 0; i < num_bins; i++) {
           bin_counts[i] = 0;
        }

        for (int i = 0; i < n; i++) {
            int bin_x = particles[i].x / bin_size;
            int bin_y = particles[i].y / bin_size;
            int bin_index = bin_y * bins_per_row + bin_x;

            if (bin_index >= 0 && bin_index < num_bins && bin_counts[bin_index] < MAX_PARTICLES_PER_BIN) {
                bins[bin_index][bin_counts[bin_index]++] = particles[i];
            }
        }

        //
        //  compute forces
        //
        for( int i = 0; i < n; i++ )
        {
            particles[i].ax = particles[i].ay = 0;
            
            int bin_x = particles[i].x / bin_size;
            int bin_y = particles[i].y / bin_size;

            for (int dx = -1; dx <= 1; dx++) {
                for (int dy = -1; dy <= 1; dy++) {
                    int nbx = bin_x + dx;
                    int nby = bin_y + dy;

                    if (nbx < 0 || nbx >= bins_per_row || nby < 0 || nby >= bins_per_row) continue;

                    int neighbor_bin = nby * bins_per_row + nbx;
                    for (int j = 0; j < bin_counts[neighbor_bin]; j++) {
                        apply_force(particles[i], bins[neighbor_bin][j]);
                    }
                }
            }
        }
        
        //
        //  move particles
        //
        for( int i = 0; i < n; i++ ) 
            move( particles[i] );
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 )
            save( fsave, n, particles );
    }
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    for (int i = 0; i < num_bins; i++) {
        free(bins[i]);
    }
    free(bins);
    free(bin_counts);
    free( particles );
    if( fsave )
        fclose( fsave );
    
    return 0;
}

